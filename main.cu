#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "DM/SistemaInicial.cuh"
#include "DM/SimulacionSinOptimizaciones.cuh"
#include "DM/SimulacionVecinos.cuh"
#include "DM/SimulacionCeldas.cuh"
#include "DM/SimulacionVecinosYCeldas.cuh"

/*
Los nombres de las variables son largos para
garantizar la claridad de lo que realizan a
cambio de legibilidad del programa.
*/

int main()
{
    /***********************************************************************/
    //Datos del programa
    std::string dir;        //directorio base
    str dpsco;              //directorio de resultados
    std::ofstream ofaedi;   //archivo de salida de datos iniciales
    std::ofstream ofapin;   //archivo de salida de posiciones Iniciales
    std::ofstream ofasres;  //archivo de salida de resultados
    std::ofstream ofasat;   //archivo de salida de posiciones de los atomos
    int maxhilos;           //numero maximo de hilos
    size_t memoria_global;  //memoria global disponible
    /***********************************************************************/
    //Datos de la corrida

    int3 condper;           //Condiciones de periodicidad
    int opt;                //optimizaciones vecinos, celdas o ambos
    int nhilos;             //cuantos hilos se encargan de realizar los 
                            //calculos para una particula
    uint nc;                //numero de configuraciones
    uint ncp;               //porcentaje de nc para el cual se calculan props
    double rc;              //radio de corte
    double dt;              //tamaño del paso de integracion
    /***********************************************************************/
    //Datos del sistema
    
    double dens;            //densidad
    double temp;            //caso NVT temperatura del baño
    double v0;              //rapidez maxima inicial de las particulas
    double3 caja;           //tamano de la caja de simulacion
    uint n_esp_m;           //numero de especies moleculares
    uint n_esp_p;           //numero de especies de particulas
    uint coord;             //indica si las coordenadas iniciales de los atomos son cartesianas o esféricas
    bool vibrante;          //determina que algoritmo de constricciones se usa, potenciales de restriccion o RATTLE
    uint *M_int;            //matriz de interaccion
    double3 celda_min;      //tamano de la celda minima
    bool p_o_m;             //nos indica si el sistema está formado por partículas(false) o moléculas(true)
    uint ensamble;          //Indica si es NVE(0) o NVT(1)
    uint termos;            //termostato RescVel(0) Andersen(1) Berendsen(2) BDP(3) NH(4)
    double param_termo;     //Cada termostato tiene 1 parametro que lo caracteriza
    /***********************************************************************/
    //Datos de moleculas
    uint nm;                //numero de moleculas
    uint *n_m_esp_mr;       //numero de moleculas de cierta especie molecular
    uint *n_p_esp_m;        //numero de particulas en cierta especie molecular
    uint max_p_en_esp_mr;   //maximo de los valores de n_p_esp_m
    uint *esp_p_en_esp_mr;  //especie de las particulas
    uint *M_int_int;        //matriz de interacciones internas
    uint *p_en_m;           //nos indica la primera particula dentro de cierta molecula 
    double kres;            //constante de union entre atomos pertenecientes a una molecula
    uint max_it;            //númmero de iteraciones máximas para el algoritmo de RATTLE
    double tol;             //toleracia para el algoritmo de RATTLE
    /***********************************************************************/
    //Datos de atomos

    uint np;
    uint nparam;
    uint *esp_p;            //arreglo con especie de cada particula
    uint3 *mad_de_p;           //particulas antes y despues de cierta particula en su molecula(Ver nota 1 antes de la inicializacion del sistema)
    uint pot;
    double *pos,*vel,*acel,*q_rat;
    double *param;
    double3 *pos_respecto_p_central;
    
    
    
    //temporales
    uint arr_temp=0;

    /*******************************************************************************************************************
    Notas
    
    Nota 1:
        Este arreglo funciona ya que al generar las moleculas sus particulas se crean consecutivamente, por ejemplo
        si la molecula m tiene 5 particulas, quiere decir que contiene las ip,ip+1,ip+2,ip+3,ip+4
        El arreglo pad_de_p funciona asi, 
            pad_de_p[ip].x a que molecula pertenece la particula
            pad_de_p[ip].y cuantas particulas hay antes de ip en la molecula
            pad_de_p[ip].z cuantas particulas hay despues de ip en la molecula
            
    Inicializacion del sistema
    *******************************************************************************************************************/
    dir="/home/gach/DMPM";
    condper=InitDataType3<int3,int>(1,1,1);
    PropiedadesGPU(maxhilos,memoria_global);
    LeerDatosSistema1(dir,n_esp_m,n_esp_p);

    n_m_esp_mr = new uint[n_esp_m];
    n_p_esp_m = new uint[n_esp_m];
    
    LeerDatosSistema2(dir,n_esp_m,n_esp_p,n_m_esp_mr,n_p_esp_m,np,nm);
    
    pos=new double[np*nd];
    q_rat=new double[np*nd];
    vel=new double[np*nd];
    acel=new double[np*nd];
    p_en_m=new uint[nm];
    int cvec=0,ccel=0;
    p_o_m = false;
    
    LeerDatosCorrida(dir,nc,ncp,coord,ensamble,termos,pot,cvec,ccel,nhilos,dt,temp,v0,rc,dens,kres,param_termo,vibrante);
    
    Nparamelec(nparam,pot);
    
    opt=cvec+2*ccel;
    param = new double[nparam*n_esp_p];
    
    LeerDatosAtomos(dir,param,pot,n_esp_p);
    
    for(int i=0;i<n_esp_m;i++)
    if(n_p_esp_m[i]>=arr_temp)arr_temp=n_p_esp_m[i];
    max_p_en_esp_mr = arr_temp;arr_temp=0;
    esp_p_en_esp_mr = new uint[max_p_en_esp_mr*n_esp_m];
    pos_respecto_p_central = new double3[max_p_en_esp_mr*n_esp_m];
    
    LeerDatosMoleculas(dir,n_esp_m,coord,n_p_esp_m,esp_p_en_esp_mr,max_p_en_esp_mr,pos_respecto_p_central);
    
    M_int = new uint[n_esp_p*n_esp_p];
    // el 2 es por que ahora solo hay 2 tipos de potenciales de restriccion: de enlace y de angulo, el numero sera 3 cuando se agregue torsion o 4 cuando agregue diedro, etc
    M_int_int = new uint[2*n_esp_m];
    
    LeerDatosInteraccion(dir,n_esp_p,M_int);
    LeerDatosInteraccionInterna(dir,n_esp_m,M_int_int);
    if(vibrante)LeerDatosRATTLE(dir,tol,max_it);
    AbrirArchivos(dir,dens,n_esp_m,n_esp_p,pot,n_m_esp_mr,ensamble,termos,nc,param_termo,param,ofaedi,ofapin,dpsco,vibrante);
    ImpresionDeDatos(nc,ncp,dt,temp,v0,rc,cvec,ccel,pot,dens,n_esp_m,n_esp_p,ensamble,termos,n_m_esp_mr,n_p_esp_m,esp_p_en_esp_mr,max_p_en_esp_mr,pos_respecto_p_central);
    ImpresionDeDatosADisco(nc,ncp,dt,temp,v0,rc,cvec,ccel,pot,dens,n_esp_m,n_esp_p,ensamble,termos,n_m_esp_mr,n_p_esp_m,esp_p_en_esp_mr,max_p_en_esp_mr,pos_respecto_p_central,ofaedi);
    celda_min=CreandoCeldaMinima(n_esp_m,pos_respecto_p_central,max_p_en_esp_mr,param,pot,esp_p_en_esp_mr,n_p_esp_m);
    
    double *centrar_m = new double[nd*n_esp_m];
    
    CentrarMoleculas(centrar_m,n_esp_m,n_p_esp_m,esp_p_en_esp_mr,max_p_en_esp_mr,pot,param,pos_respecto_p_central);
    
    esp_p = new uint[np];
    mad_de_p = new uint3[np];
    
    ConfiguracionCubica(n_esp_m,n_m_esp_mr,n_p_esp_m,p_en_m,pos,pos_respecto_p_central,max_p_en_esp_mr,caja,centrar_m,celda_min,dens,ofapin,esp_p_en_esp_mr,nm,esp_p,mad_de_p);
    
    double3 cajai=InvDataType3<double3>(caja);
    double *dis_p_esp_mr_rep = new double[n_esp_m*max_p_en_esp_mr*max_p_en_esp_mr];
    if(max_p_en_esp_mr>1)p_o_m=true;
    
    InicializarVelocidades(v0,vel,np);
    if(!p_o_m)DistanciasEntreParticulasEnMoleculaIniciales(np,n_esp_m,max_p_en_esp_mr,n_p_esp_m,dis_p_esp_mr_rep,pos_respecto_p_central);
    /*******************************************************************************************************************/
    //Lo usamos para optimizaciones
    double rbuf=0.5;
    ArchivosDeResultados(dpsco,ofasres,ofasat,opt);
    
    switch(opt)
    {
        case 0:
        Simulacion(nc,ncp,nhilos,np,nparam,pot,n_esp_p,n_esp_m,max_p_en_esp_mr,ensamble,termos,max_it,mad_de_p,esp_p,
                   n_m_esp_mr,n_p_esp_m,M_int,p_en_m,maxhilos,vibrante,memoria_global,dt,dens,kres,temp,param_termo,tol,param,
                   pos,q_rat,vel,acel,dis_p_esp_mr_rep,condper,caja,cajai,ofasres,ofasat);
        break;
        case 1:
        SimulacionV(nc,ncp,np,n_esp_p,n_esp_m,nparam,pot,max_p_en_esp_mr,ensamble,termos,max_it,esp_p,M_int,p_en_m,
                    n_m_esp_mr,n_p_esp_m,nhilos,maxhilos,vibrante,mad_de_p,condper,rc,rbuf,dens,dt,kres,temp,param_termo,tol,
                    param,pos,vel,acel,q_rat,dis_p_esp_mr_rep,caja,cajai,memoria_global,ofasres,ofasat);
        break;
        case 2:
        SimulacionC(nc,ncp,np,n_esp_p,n_esp_m,nparam,pot,max_p_en_esp_mr,ensamble,termos,max_it,esp_p,M_int,p_en_m,
                    n_m_esp_mr,n_p_esp_m,nhilos,maxhilos,vibrante,rc,dt,dens,kres,temp,param_termo,tol,param,pos,vel,acel,
                    q_rat,mad_de_p,condper,caja,cajai,dis_p_esp_mr_rep,memoria_global,ofasat,ofasres);
        break;
        case 3:
        SimulacionVYC(nc,ncp,np,n_esp_p,n_esp_m,nparam,pot,max_p_en_esp_mr,ensamble,termos,max_it,esp_p,M_int,p_en_m,
                      n_m_esp_mr,n_p_esp_m,nhilos,maxhilos,vibrante,rc,rbuf,dens,dt,kres,temp,param_termo,tol,param,pos,vel,acel,
                      q_rat,dis_p_esp_mr_rep,mad_de_p,condper,caja,cajai,memoria_global,ofasres,ofasat);
        break;
    }
    
    delete[] pos;
    delete[] vel;
    delete[] acel;
    return 0;
}
