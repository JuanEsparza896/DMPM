#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "SistemaInicial.hpp"
#include "OperacionesTDatosCuda.cuh"
#include "OperacionesDeHilosyBloques.cuh"
#include "SimulacionSinOptimizaciones.cuh"
#include "SimulacionVecinos.cuh"
#include "SimulacionCeldas.cuh"
#include "SimulacionVecinosYCeldas.cuh"


int main()
{
    /***********************************************************************/
    //Datos del programa
    std::string dir;        //directorio base
    str dpsco;              //directorio de resultados
    std::ofstream ofaedi;   //archivo de salida de datos iniciales
    std::ofstream ofapin;   //archivo de salida de posiciones Iniciales
    std::ofstream ofasres;  //archivo de salida de resultados
    std::ofstream ofasat;   //archivo de salida de posiciones de los atomos
    int maxhilos;           //numero maximo de hilos
    size_t memoria_global;  //memoria global disponible
    /***********************************************************************/
    //Datos de la corrida

    int3 condper;           //Condiciones de periodicidad
    int opt;                //optimizaciones vecinos, celdas o ambos
    int nhilos;             //cuantos hilos se encargan de realizar los 
                            //calculos para una particula
    uint nc;                //numero de configuraciones
    uint ncp;               //porcentaje de nc para el cual se calculan props
    double rc;              //radio de corte
    double dt;              //tamaño del paso de integracion
    /***********************************************************************/
    //Datos del sistema
    
    double dens;            //densidad
    double temp;            //caso NVT temperatura del baño
    double v0;              //rapidez maxima inicial de las particulas
    double3 caja;           //tamano de la caja de simulacion
    uint n_esp_m;           //numero de especies moleculares
    uint n_esp_p;           //numero de especies de particulas
    bool vibrante;          //determina que algoritmo de constricciones se usa, potenciales de restriccion o RATTLE
    uint *M_int;            //matriz de interaccion
    double3 celda_min;      //tamano de la celda minima
    /***********************************************************************/
    //Datos de moleculas
    uint nm;                //numero de moleculas
    uint *n_m_esp_mr;       //numero de moleculas de cierta especie molecular
    uint *n_p_esp_m;        //numero de particulas en cierta especie molecular
    uint max_p_en_esp_mr;   //maximo de los valores de n_p_esp_m
    uint *esp_p_en_esp_mr;  //especie de las particulas
    uint *M_int_int;        //matriz de interacciones internas
    uint *p_en_m;           //nos indica la primera particula dentro de cierta molecula 
    /***********************************************************************/
    //Datos de atomos

    uint np;
    uint nparam;
    uint *esp_p;            //arreglo con especie de cada particula
    uint3 *mad_de_p;           //particulas antes y despues de cierta particula en su molecula(Ver nota 1 antes de la inicializacion del sistema)
    uint pot;
    double *pos,*vel,*acel;
    double *param;
    double3 *pos_respecto_p_central;
    
    
    
    //temporales
    uint arr_temp=0;

    /*******************************************************************************************************************
    Notas
    
    Nota 1:
        Este arreglo funciona ya que al generar las moleculas sus particulas se crean consecutivamente, por ejemplo
        si la molecula m tiene 5 particulas, quiere decir que contiene las ip,ip+1,ip+2,ip+3,ip+4
        El arreglo pad_de_p funciona asi, 
            pad_de_p[ip].x a que molecula pertenece la particula
            pad_de_p[ip].y cuantas particulas hay antes de ip en la molecula
            pad_de_p[ip].z cuantas particulas hay despues de ip en la molecula
            
    Inicializacion del sistema
    *******************************************************************************************************************/
    dir="/home/gach/DMPM";
    condper=InitDataType3<int3>(1,1,1);
    PropiedadesGPU(maxhilos,memoria_global);
    LeerDatosSistema1(dir,n_esp_m,n_esp_p);
    n_m_esp_mr = new uint[n_esp_m];
    n_p_esp_m = new uint[n_esp_m];
    LeerDatosSistema2(dir,n_esp_m,n_esp_p,n_m_esp_mr,n_p_esp_m,np,nm);
    pos=new double[np*nd];
    vel=new double[np*nd];
    acel=new double[np*nd];
    p_en_m=new uint[nm];
    int cvec=0,ccel=0;
    LeerDatosCorrida(dir,nc,ncp,dt,temp,v0,rc,dens,pot,cvec,ccel,nhilos,vibrante,nparam);
    opt=cvec+2*ccel;
    param = new double[nparam*n_esp_p];
    LeerDatosAtomos(dir,param,nparam,n_esp_p);
    for(int i=0;i<n_esp_m;i++)
    if(n_p_esp_m[i]>=arr_temp)arr_temp=n_p_esp_m[i];
    max_p_en_esp_mr = arr_temp;arr_temp=0;
    esp_p_en_esp_mr = new uint[max_p_en_esp_mr*n_esp_m];
    pos_respecto_p_central = new double3[max_p_en_esp_mr*n_esp_m];
    LeerDatosMoleculas(dir,n_esp_m,n_p_esp_m,esp_p_en_esp_mr,max_p_en_esp_mr,pos_respecto_p_central);
    M_int = new uint[n_esp_p*n_esp_p];
    // el 2 es por que ahora solo hay 2 tipos de potenciales de restriccion: de enlace y de angulo, el numero sera 3 cuando se agregue torsion o 4 cuando agregue diedro, etc
    M_int_int = new uint[2*n_esp_m];
    LeerDatosInteraccion(dir,n_esp_p,M_int);
    LeerDatosInteraccionInterna(dir,n_esp_m,M_int_int);
    AbrirArchivos(dir,dens,n_esp_m,n_esp_p,n_m_esp_mr,nparam,param,ofaedi,ofapin,dpsco);
    ImpresionDeDatos(nc,ncp,dt,temp,v0,rc,cvec,ccel,pot,dens,n_esp_m,n_esp_p,n_m_esp_mr,n_p_esp_m,esp_p_en_esp_mr,max_p_en_esp_mr,pos_respecto_p_central);
    ImpresionDeDatosADisco(nc,ncp,dt,temp,v0,rc,cvec,ccel,pot,dens,n_esp_m,n_esp_p,n_m_esp_mr,n_p_esp_m,esp_p_en_esp_mr,max_p_en_esp_mr,pos_respecto_p_central,ofaedi);
    celda_min=CreandoCeldaMinima(n_esp_m,pos_respecto_p_central,max_p_en_esp_mr,param,nparam,esp_p_en_esp_mr,n_p_esp_m);
    double *centrar_m = new double[nd*n_esp_m];
    CentrarMoleculas(centrar_m,n_esp_m,n_p_esp_m,esp_p_en_esp_mr,max_p_en_esp_mr,nparam,param,pos_respecto_p_central);
    esp_p = new uint[np];
    mad_de_p = new uint3[np];
    ConfiguracionCubica(n_esp_m,n_m_esp_mr,n_p_esp_m,p_en_m,pos,pos_respecto_p_central,max_p_en_esp_mr,caja,centrar_m,celda_min,dens,ofapin,esp_p_en_esp_mr,nm,esp_p,mad_de_p);
    double3 cajai=InvDataType3<double3>(caja);
    InicializarVelocidades(v0,vel,np);
    /*******************************************************************************************************************/
    //Lo usamos para optimizaciones
    double rbuf=0.5;
    ArchivosDeResultados(dpsco,ofasres,ofasat,opt);
    
    switch(opt)
    {
        case 0:
        Simulacion(nc,ncp,nhilos,np,nparam,pot,n_esp_p,mad_de_p,esp_p,M_int,maxhilos,memoria_global,dt,dens,param,pos,vel,acel,condper,caja,cajai,ofasres,ofasat);
        break;
        case 1:
        SimulacionV(nc,ncp,np,n_esp_p,nparam,pot,esp_p,M_int,nhilos,maxhilos,mad_de_p,condper,rc,rbuf,dens,temp,dt,param,pos,vel,acel,caja,cajai,memoria_global,ofasres,ofasat);
        break;
        case 2:
        SimulacionC(nc,ncp,np,n_esp_p,nparam,pot,esp_p,M_int,nhilos,maxhilos,rc,temp,dt,dens,param,pos,vel,acel,mad_de_p,condper,caja,cajai,memoria_global,ofasat,ofasres);
        break;
        case 3:
        SimulacionVYC(nc,ncp,np,n_esp_p,nparam,pot,esp_p,M_int,nhilos,maxhilos,rc,rbuf,dens,temp,dt,param,pos,vel,acel,mad_de_p,condper,caja,cajai,memoria_global,ofasres,ofasat);
        break;
    }
    
    delete[] pos;
    delete[] vel;
    delete[] acel;
    return 0;
}
