#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "SistemaInicial.hpp"
#include "OperacionesTDatosCuda.cuh"
#include "SimulacionSinOptimizaciones.cuh"
#include "SimulacionVecinos.cuh"
#include "SimulacionCeldas.cuh"
#include "SimulacionVecinosYCeldas.cuh"

int main()
{
    /***********************************************************************/
    //Datos del programa
    std::string dir;        //directorio base
    str dpsco;              //directorio de resultados
    std::ofstream ofaedi;   //archivo de salida de datos iniciales
    std::ofstream ofapin;   //archivo de salida de posiciones Iniciales
    std::ofstream ofasres;  //archivo de salida de resultados
    std::ofstream ofasat;   //archivo de salida de posiciones de los atomos
    int maxhilos;           //numero maximo de hilos
    size_t memoria_global;  //memoria global disponible
    /***********************************************************************/
    //Datos de la corrida

    int3 condper;           //Condiciones de periodicidad
    int opt;                //optimizaciones vecinos, celdas o ambos
    int nhilos;             //cuantos hilos se encargan de realizar los 
                            //calculos para una particula
    uint nc;                //numero de configuraciones
    uint ncp;               //porcentaje de nc para el cual se calculan props
    double rc;              //radio de corte
    double dt;              //tamaño del paso de integracion
    /***********************************************************************/
    //Datos del sistema
    
    double dens;            //densidad
    double temp;            //caso NVT temperatura del baño
    double v0;              //rapidez maxima inicial de las particulas
    double3 caja;           //tamano de la caja de simulacion
    int nd;                 //dimension en la que se trabaja
    /***********************************************************************/
    //Datos de atomos

    uint np;
    //pot ya no sirve, ahora necesitamos la matriz de interacciones
    //np necesita no ser dato en el documento de texto, viene de 
    //eps y sig varian dependiendo de la especie atomica 
    int pot;
    double *p,*v,*a;
    double eps,sig;
    
    
    



    //Inicializacion del sistema
    /*******************************************************************************************************************/
    dir="/home/gach/DMPM";
    condper=InitDataType3<int3>(1,1,1);
    PropiedadesGPU(maxhilos,memoria_global);
    LeerDatos(dir,dens,nd,np,nc,ncp,dt,temp,v0,rc,pot,eps,sig,dpsco,ofaedi,ofapin,opt,nhilos);
    p=new double[np*nd];
    v=new double[np*nd];
    a=new double[np*nd];
    Cuadrada(np,nd,sig,caja,dens,ofapin,p);
    double3 dcajai=InvDataType3<double3>(caja);
    ImprimirDatos(dens,nd,np,pot,sig,eps,caja,nc,ncp,dt,rc,ofaedi);
    VelocidadesInicialesalAzar(v0,v,np,nd);
    /*******************************************************************************************************************/
    double rbuf=0.5;
    ArchivosDeResultados(dpsco,ofasres,ofasat,opt);
    switch(opt)
    {
        case 0:
        
        Simulacion(np,nd,p,v,a,sig,eps,caja,dcajai,condper,temp,ofasres,ofasat,nc,dt,dens,ncp,nhilos,pot,maxhilos,memoria_global);
        break;
        case 1:
        SimulacionV(np,nd,p,v,a,sig,eps,caja,dcajai,condper,temp,ofasres,ofasat,nc,dt,dens,ncp,rc,rbuf,nhilos,pot,maxhilos,memoria_global);
        break;
        case 2:
        SimulacionC(np,nd,p,v,a,sig,eps,caja,dcajai,condper,temp,ofasres,ofasat,nc,dt,dens,ncp,rc,rbuf,nhilos,pot,maxhilos,memoria_global);
        break;
        case 3:
        SimulacionVYC(np,nd,p,v,a,sig,eps,caja,dcajai,condper,temp,ofasres,ofasat,nc,dt,dens,ncp,rc,rbuf,nhilos,pot,maxhilos,memoria_global);
        break;
    }
    delete[] p;
    delete[] v;
    delete[] a;
    return 0;
}
