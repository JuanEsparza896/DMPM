#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "SistemaInicial.hpp"
#include "OperacionesTDatosCuda.cuh"
#include "SimulacionSinOptimizaciones.cuh"
#include "SimulacionVecinos.cuh"
#include "SimulacionCeldas.cuh"
#include "SimulacionVecinosYCeldas.cuh"

int main()
{
    std::string dir="/home/gach/DMP2";
    str dpsco;
    int3 condper=InitDataType3<int3>(1,1,1);
    double dens,dt,temp,v0,rc,eps,sig,cajax,cajay,cajaz;
    int nd,np,nc,ncp,pot,opt,nhilos;
    double *p,*v,*a;
    std::ofstream ofaedi,ofapin,ofasres,ofasat;
    int maxhilos;
    size_t memoria_global;
    PropiedadesGPU(maxhilos,memoria_global);

    //Inicializacion del sistema
    /*******************************************************************************************************************/
    LeerDatos(dir,dens,nd,np,nc,ncp,dt,temp,v0,rc,pot,eps,sig,dpsco,ofaedi,ofapin,opt,nhilos);
    p=new double[np*nd];
    v=new double[np*nd];
    a=new double[np*nd];
    Cuadrada(np,nd,sig,cajax,cajay,cajaz,dens,ofapin,p);
    double3 dcaja=InitDataType3<double3>(cajax,cajay,cajaz);
    double3 dcajai=InvDataType3<double3>(dcaja);
    ImprimirDatos(dens,nd,np,pot,sig,eps,cajax,cajay,cajaz,nc,ncp,dt,rc,ofaedi);
    VelocidadesInicialesalAzar(v0,v,np,nd);
    /*******************************************************************************************************************/
    double rbuf=0.5;
    switch(opt)
    {
        case 0:
        printf("\nNo se usan optimizaciones\n");
        ArchivosDeResultados(dpsco,ofasres,ofasat,"SinOptimizaciones");
        Simulacion(np,nd,p,v,a,sig,eps,dcaja,dcajai,condper,temp,ofasres,ofasat,nc,dt,dens,ncp,nhilos,pot,maxhilos,memoria_global);
        break;
        case 1:
        printf("\nOptimizaciones: Vecinos\n");
        ArchivosDeResultados(dpsco,ofasres,ofasat,"Vecinos");
        SimulacionV(np,nd,p,v,a,sig,eps,dcaja,dcajai,condper,temp,ofasres,ofasat,nc,dt,dens,ncp,rc,rbuf,nhilos,pot,maxhilos,memoria_global);
        break;
        case 2:
        printf("\nOptimizaciones: Celdas\n");
        ArchivosDeResultados(dpsco,ofasres,ofasat,"Celdas");
        SimulacionC(np,nd,p,v,a,sig,eps,dcaja,dcajai,condper,temp,ofasres,ofasat,nc,dt,dens,ncp,rc,rbuf,nhilos,pot,maxhilos,memoria_global);
        break;
        case 3:
        printf("\nOptimizaciones: Vecinosy Celdas\n");
        ArchivosDeResultados(dpsco,ofasres,ofasat,"Vecinos_Celdas");
        SimulacionVYC(np,nd,p,v,a,sig,eps,dcaja,dcajai,condper,temp,ofasres,ofasat,nc,dt,dens,ncp,rc,rbuf,nhilos,pot,maxhilos,memoria_global);
        break;
    }
    delete[] p;
    delete[] v;
    delete[] a;
    return 0;
}
